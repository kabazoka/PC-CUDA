#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 100000000

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x; // this thread handles the data at its thread index
    for (int i = tid; i < N; i += blockDim.x) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    struct timespec t_start, t_end;
    double elapsedTimeCPU;

    // Start CPU timing
    clock_gettime(CLOCK_REALTIME, &t_start);

    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    // Allocate memory on the CPU
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));

    // Allocate memory on the GPU
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // Fill the arrays 'a' and 'b' on the CPU
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 256;
        b[i] = rand() % 256;
    }

    // Copy arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // CUDA event creation and start recording
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // GPU kernel function
    add<<<1, 1024>>>(dev_a, dev_b, dev_c);

    // CUDA event end and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Compute execution time for GPU
    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, start, stop);

    // Copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify the GPU computation
    bool success = true;
    for (int i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success) {
        printf("We did it!\n");
    }

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Stop CPU timing
    clock_gettime(CLOCK_REALTIME, &t_end);

    // Compute and print the elapsed time in millisec for CPU
    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;

    printf("GPU time: %f msec\n", elapsedTimeGPU);
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);

    return 0;
}
