#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 100000000
__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main(void) {
    struct timespec t_start, t_end;
    double elapsedTimeCPU;
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    // Start CPU timing
    clock_gettime(CLOCK_REALTIME, &t_start);

    // allocate the memory on the CPU
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));

    // allocate the memory on the GPU
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 256;
        b[i] = rand() % 256;
    }
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // CUDA event creation and start recording
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // CUDA event end and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Compute execution time for GPU
    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, start, stop);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success)
        printf("We did it!\n");

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Stop CPU timing
    clock_gettime(CLOCK_REALTIME, &t_end);

    // Compute and print the elapsed time in millisec for CPU
    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;

    printf("GPU time: %f msec\n", elapsedTimeGPU);
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);

    return 0;
}
